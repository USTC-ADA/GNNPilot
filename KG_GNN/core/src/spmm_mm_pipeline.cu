#include "hip/hip_runtime.h"
#include "../include/KG_GNN.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#include <cuda_pipeline.h>
// #include <hip/hip_cooperative_groups.h>
// #include <cooperative_groups/memcpy_async.h>

// namespace cg = cooperative_groups;

#define MM_MEM_PER_BLOCK 4
#define MM_CALC_PER_BLOCK 4
#define SPMM_PER_BLOCK 16

#define MINDEX(d, i, j, m, n) ((d) * (m) * (n) + (i) * (n) + (j))

// l: large dimension s: small dimension
// e.g. ls indicates an m x n matrix where m is large and n is small

#define M_TILE 8
#define K_TILE 32
#define N_TILE 32
#define K_TILE_SM (K_TILE)
#define N_TILE_SM (N_TILE)
#define M_TILE_BLOCK (MM_CALC_PER_BLOCK * M_TILE)

__global__ void kg_spmm_ls_ss_mm_pipeline(int m, int n, int k, int *rowptr, int *colidx, float *input_a, float *input_b, float *output_c)
{
    extern __shared__ float s[];
    float *a_tile = (float*)&s[0];
    float *b_tile = (float*)&s[(MM_CALC_PER_BLOCK * M_TILE * K_TILE_SM) * 2];
    float *c_tile = (float*)&s[(MM_CALC_PER_BLOCK * M_TILE * K_TILE_SM + K_TILE * N_TILE_SM) * 2];

    // cg::thread_block tb = cg::this_thread_block();
    if (threadIdx.x < SPMM_PER_BLOCK * WARP_SIZE)
    {
        int local_tid = threadIdx.x;
        int local_wid = local_tid / WARP_SIZE;
        int lane_id = local_tid & (WARP_SIZE - 1);

        int global_tid = blockIdx.x * MM_MEM_PER_BLOCK * WARP_SIZE + local_tid;
        int global_wid = global_tid / WARP_SIZE;

        int count = 0;

        for (int n_block = 0; n_block < n; n_block += N_TILE)
        {
            if (n_block == 0)
            {
                // K tile iteration
                for (int k_block = 0; k_block < k; k_block += K_TILE)
                {
                    int dbuff_i = count % 2;

                    int row_st = blockIdx.x * M_TILE_BLOCK;
                    int row_ed = (blockIdx.x + 1) * M_TILE_BLOCK;
                    if (row_ed >= m) row_ed = m;

                    // naive spmm, each warp calculate one row
                    for (int row = row_st + local_wid; row < row_ed; row += SPMM_PER_BLOCK)
                    {
                        float result[K_TILE / WARP_SIZE];
                        #pragma unroll
                        for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                            result[kk / WARP_SIZE] = 0;
                        float degree_inv = 1.0;

                        //int idx_ed = rowptr[row + 1];
                        //if (idx_ed > rowptr[row] + 32) idx_ed = rowptr[row] + 32;
                        //for (int idx = rowptr[row]; idx < idx_ed; idx++)
                        for (int idx = rowptr[row]; idx < rowptr[row + 1]; idx++)
                        {
                            int col = colidx[idx];
                            #pragma unroll
                            for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                            {
                                result[kk / WARP_SIZE] += input_a[col * k + k_block + kk + lane_id] * degree_inv;
                                // if (local_tid == 0) printf("col %d input_a %.4f\n", colidx[idx], input_a[col * k + k_block + kk + lane_id]);
                            }
                        }
                        // if (row == 31 && !lane_id) printf("row %d res: ", row);
                        // for (int debug_i = 0; debug_i < WARP_SIZE; debug_i++)
                        //     if (row == 31 && debug_i == lane_id) printf("%.4f ", result[0]);
                        // if (row == 31 && !lane_id) printf("\n");

                        // if (row == 0 && lane_id == 31) printf("row %d res: %.4f\n", row, result[0]);

                        #pragma unroll
                        for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                            a_tile[MINDEX(dbuff_i, row - row_st, kk * WARP_SIZE + lane_id, M_TILE_BLOCK, K_TILE_SM)] = 
                            result[kk / WARP_SIZE];
                    }

                    // nnz balanced spmm
                    // int block_nnz = (rowptr[row_ed] - rowptr[row_st] + SPMM_PER_BLOCK - 1) / SPMM_PER_BLOCK;

                    // int idx_st = rowptr[row_st] + block_nnz * local_wid;
                    // int idx_ed = rowptr[row_st] + block_nnz * (local_wid + 1);
                    // if (idx_ed > rowptr[row_ed])
                    //     idx_ed = rowptr[row_ed];

                    // // if (local_wid == 1) printf("%d idx_st: %d idx_ed: %d\n", lane_id, idx_st, idx_ed);

                    // int row_i = row_st;

                    // float result[K_TILE / WARP_SIZE];
                    // float degree_inv = 1.0;

                    // #pragma unroll
                    // for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                    //     result[kk / WARP_SIZE] = 0;

                    // for (int row = row_st; row < row_ed; row++)
                    //     if (rowptr[row] >= idx_st)
                    //     {
                    //         row_i = row;
                    //         break;
                    //     }

                    // int idx = idx_st;
                    // while (idx < idx_ed)
                    // {
                    //     int col = colidx[idx];
                    //     if (idx == rowptr[row_i + 1])
                    //     {
                    //         #pragma unroll
                    //         for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                    //         {
                    //             atomicAdd(&a_tile[MINDEX(dbuff_i, row_i - row_st, kk * WARP_SIZE + lane_id, M_TILE_BLOCK, K_TILE_SM)],
                    //             result[kk / WARP_SIZE]);
                    //             //if (lane_id == 0) printf("%d:( %d, %d, %.4f)\n", local_wid, row_i, idx_st, result[kk / WARP_SIZE]);
                    //             result[kk / WARP_SIZE] = 0;
                    //         }
                    //         row_i++;
                    //     }
                    //     #pragma unroll
                    //     for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                    //         result[kk / WARP_SIZE] += input_a[col * k + k_block + kk + lane_id] * degree_inv;

                    //     idx++;
                    // }

                    // float result[K_TILE / WARP_SIZE * M_TILE_BLOCK];
                    // #pragma unroll
                    // for (int mm = 0; mm < M_TILE_BLOCK; mm++)
                    //     for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                    //         result[mm * kk / WARP_SIZE + kk] = 0;
                    // for (int idx = idx_st; idx < idx_ed; idx++)
                    // {
                    //     int tgt_reg = 
                    //     for (int kk = 0; kk < K_TILE; kk+=WARP_SIZE)
                    //     {
                    //         int tgt_reg =  + kk;
                    //     }
                    // }

                    count++;

                    __syncthreads();
                }
            }
        }
    }
    else
    {
        int local_tid = (threadIdx.x - SPMM_PER_BLOCK * WARP_SIZE) % (MM_MEM_PER_BLOCK * WARP_SIZE);
        int local_wid = local_tid / WARP_SIZE;
        int lane_id = local_tid & (WARP_SIZE - 1);

        int global_tid = blockIdx.x * MM_MEM_PER_BLOCK * WARP_SIZE + local_tid;
        int global_wid = global_tid / WARP_SIZE;

        #define MT_TILE (M_TILE / 4)
        #define NT_TILE (N_TILE / 8)
        //#define NT_TILE ((M_TILE * N_TILE) / WARP_SIZE / MT_TILE)
        #define KT_TILE 4

        int nt_tiles = N_TILE / NT_TILE;

        int nt_tile_idx = (local_tid % nt_tiles) * NT_TILE;
        int mt_tile_idx = (local_tid / nt_tiles) * MT_TILE;

        int input_a_st = global_wid * M_TILE;
        int m_local_st = local_wid * M_TILE;

        if (threadIdx.x < (MM_MEM_PER_BLOCK + SPMM_PER_BLOCK) * WARP_SIZE)
        {
            //if (local_tid == 0 && lane_id == 0) printf("threadIdx: %d\n", threadIdx.x);

            int count = 0;
            // int ln_block = -1, lk_block = -1;
            // int lln_block = -1, llk_block = -1;

            bool zfill = false;

            if (lane_id % 4 == 0) zfill = true;
            int src_in_bytes = 0;

            for (int n_block = 0; n_block < n; n_block += N_TILE)
            {
                // K tile iteration
                for (int k_block = 0; k_block < k; k_block += K_TILE)
                {
                    int dbuff_i = count % 2;

                    int j_st = lane_id % 8;
                    int i_st = lane_id / 8;

                    // read in B tile
                    #pragma unroll
                    for (int i = 0; i < K_TILE / MM_MEM_PER_BLOCK; i++)
                    {
                        int tgt_i = i + local_wid * K_TILE / MM_MEM_PER_BLOCK;
                        //for (int j = lane_id; j < N_TILE; j+=WARP_SIZE)
                        #pragma unroll
                        for (int j = 0; j < N_TILE / WARP_SIZE; j++)
                        {
                            int tgt_j = j * WARP_SIZE + lane_id;
                            b_tile[MINDEX(dbuff_i, tgt_i, tgt_j, K_TILE, N_TILE)] = 
                            input_b[(k_block + tgt_i) * n + n_block + tgt_j];
                            // cg::memcpy_async(tb, &b_tile[MINDEX(dbuff_i, tgt_i, tgt_j, K_TILE, N_TILE)], 
                            // &input_b[(k_block + tgt_i) * n + n_block + tgt_j], sizeof(float));
                            // // __pipeline_memcpy_async(&b_tile[MINDEX(dbuff_i, tgt_i, tgt_j, K_TILE, N_TILE_SM)],
                            // &input_b[(k_block + tgt_i) * n + n_block + tgt_j], sizeof(float));
                        }
                    } 

                    // __pipeline_commit();
                    // __pipeline_wait_prior(0);

                    asm volatile("cp.async.wait_all;\n" ::);

                    // asm volatile("cp.async.commit_group;\n" ::);
                    // asm volatile("cp.async.wait_group %0;\n" :: "n"(0));

                    count++;

                    //dbuff_i = (dbuff_i + 1) % 2;
                    __syncthreads();
                }

                // // write back last C tile
                // if (count >= 2)
                // {
                //     for (int i = 0; i < M_TILE; i++)
                //         for (int j = lane_id; j < N_TILE; j+=WARP_SIZE)
                //             output_c[input_a_st * n + i * n + n_block + j] = c_tile[MINDEX(dbuff_i, m_local_st + i, j, M_TILE_BLOCK, N_TILE)];
                // }
            }
        }
        else if (threadIdx.x < (SPMM_PER_BLOCK + MM_MEM_PER_BLOCK + MM_CALC_PER_BLOCK) * WARP_SIZE)
        {
            int count = 0;
            //if (local_tid == 0 && lane_id == 0) printf("threadIdx: %d\n", threadIdx.x);

            for (int n_block = 0; n_block < n; n_block += N_TILE)
            {
                // float a_thread_tile[MT_TILE][KT_TILE];
                // float b_thread_tile[KT_TILE][NT_TILE];
                float c_thread_tile[MT_TILE][NT_TILE];

                for (int i = 0; i < MT_TILE; i++)
                    for (int j = 0; j < NT_TILE; j++)
                        c_thread_tile[i][j] = 0;

                // K tile iteration
                #pragma unroll
                for (int k_block = 0; k_block < k; k_block += K_TILE)
                {
                    //asm volatile("bar.sync %0, %1;" : : "r"(local_wid), "r"(64) : "memory");
                    __syncthreads();

                    int dbuff_i = count % 2;

                    // #pragma unroll
                    // for (int k = 0; k < K_TILE; k += KT_TILE)
                    // {
                    //     for (int i = 0; i < MT_TILE; i++)
                    //         for (int kk = 0; kk < KT_TILE; kk++)
                    //             //a_thread_tile[i][kk] = a_tile[(mt_tile_idx + i) * K_TILE + k + kk];
                    //             a_thread_tile[i][kk] = a_tile[MINDEX(dbuff_i, mt_tile_idx + i, k + kk, M_TILE_BLOCK, K_TILE_SM)];
                    //     for (int kk = 0; kk < KT_TILE; kk++)
                    //         for (int j = 0; j < NT_TILE; j++)
                    //             //b_thread_tile[kk][j] = b_tile[(k + kk) * N_TILE + nt_tile_idx + j];
                    //             b_thread_tile[kk][j] = b_tile[MINDEX(dbuff_i, k + kk, nt_tile_idx + j, K_TILE, N_TILE_SM)];

                    //     for (int i = 0; i < MT_TILE; i++)
                    //         for (int j = 0; j < NT_TILE; j++)
                    //             for (int kk = 0; kk < KT_TILE; kk++)
                    //                 c_thread_tile[i][j] += a_thread_tile[i][kk] * b_thread_tile[kk][j];
                    //             //c_thread_tile[i][j] += a_thread_tile[i][0] * b_thread_tile[0][j];
                    // }

                    float a_thread_tile[MT_TILE];
                    float b_thread_tile[NT_TILE];

                    #pragma unroll
                    for (int k = 0; k < K_TILE; k++)
                    {
                        #pragma unroll
                        for (int i = 0; i < MT_TILE; i++)
                            a_thread_tile[i] = a_tile[MINDEX(dbuff_i, mt_tile_idx + i, k, M_TILE_BLOCK, K_TILE_SM)];

                        #pragma unroll
                        for (int j = 0; j < NT_TILE; j++)
                            b_thread_tile[j] = b_tile[MINDEX(dbuff_i, k, nt_tile_idx + j, K_TILE, N_TILE_SM)];

                        #pragma unroll
                        for (int i = 0; i < MT_TILE; i++)
                            #pragma unroll
                            for (int j = 0; j < NT_TILE; j++)
                                //c_thread_tile[i][j] += a_thread_tile[i] * b_thread_tile[j];
                                c_thread_tile[i][j] = fmaf(a_thread_tile[i], b_thread_tile[j], c_thread_tile[i][j]); 
                    }

                    // if (local_tid == 0 && lane_id == 0)
                    // {
                    //     printf("a_thread_tile, %.4f\n", a_thread_tile[0]);
                    //     printf("b_thread_tile, %.4f\n", b_thread_tile[0]);
                    //     printf("c_thread_tile, %.4f\n", c_thread_tile[0][0]);
                    // }
                    count++;
                    //dbuff_i = (dbuff_i + 1) % 2;

                }

                for (int i = 0; i < MT_TILE; i++)
                    for (int j = 0; j < NT_TILE; j++)
                        //c_tile[(mt_tile_idx + i) * N_TILE + nt_tile_idx + j] = c_thread_tile[i][j];
                        c_tile[MINDEX(0, mt_tile_idx + i, nt_tile_idx + j, M_TILE_BLOCK, N_TILE_SM)] = c_thread_tile[i][j];

                //write back C tile
                for (int i = 0; i < M_TILE; i++)
                    if (input_a_st + i < m)
                        for (int j = lane_id; j < N_TILE; j+=WARP_SIZE)
                            output_c[input_a_st * n + i * n + n_block + j] = c_tile[MINDEX(0, m_local_st + i, j, M_TILE_BLOCK, N_TILE_SM)];

            }
        }
    }

}

#undef MT_TILE
#undef NT_TILE
#undef KT_TILE

void kg_spmm_mm_pipeline_execute(int m, int n, int k, int *rowptr, int *colidx, float *input_a, float *input_b, float *output_c)
{
    dim3 thread_num((SPMM_PER_BLOCK + MM_MEM_PER_BLOCK + MM_CALC_PER_BLOCK) * WARP_SIZE);
    // printf("%d", thread_num);
    int mtile_pb = M_TILE * MM_CALC_PER_BLOCK;
    dim3 block_num((m + mtile_pb - 1) / mtile_pb);

    int shared_size = (2 * (MM_CALC_PER_BLOCK * M_TILE * K_TILE_SM 
    + K_TILE * N_TILE_SM) + MM_CALC_PER_BLOCK * M_TILE * N_TILE_SM) * sizeof(float);

    //printf("%d\n", shared_size);

    //printf("mtile_pb %d block_num %d\n", mtile_pb, (m + mtile_pb - 1) / mtile_pb);

    hipFuncSetAttribute(reinterpret_cast<const void*>(kg_spmm_ls_ss_mm_pipeline), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size);

    kg_spmm_ls_ss_mm_pipeline<<<block_num, thread_num, shared_size>>>(m, n, k, rowptr, colidx, input_a, input_b, output_c);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
}

#undef M_TILE
#undef N_TILE
#undef K_TILE
